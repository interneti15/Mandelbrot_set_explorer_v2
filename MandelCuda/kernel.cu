#include "hip/hip_runtime.h"
﻿#include <thread>
#include <iostream>

#include <SFML/Graphics.hpp>
#include <boost/multiprecision/cpp_dec_float.hpp>
#include <boost/multiprecision/cpp_int.hpp>
#include <hip/hip_runtime.h>

//#include "classes.hpp"
#include "functions.hpp"

#include <mutex>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>


using namespace boost::multiprecision;
using namespace std;

void end(globals& Global, const int& code, threadsHandling& Threads, thread* SC)
{
	Threads.killAll(&Global);

	Global.Pend = true;
	SC->join();

	delete[] Global.screen;
	delete[] Global.pixels;

	exit(code);
}

void cpTest() {
	constexpr int size = 50;

	myNumLib::precisionNumber A = myNumLib::precisionNumber::precisionNumberConstructor(size);
	myNumLib::precisionNumber B = myNumLib::precisionNumber::precisionNumberConstructor(size);
	myNumLib::precisionNumber C = myNumLib::precisionNumber::precisionNumberConstructor(size);

	for (size_t i = 0; i < size; i++)
	{
		A.top.number[i] = 1;
		B.top.number[i] = 2;
	}

	C.top = myNumLib::bigInt::add(A.top, B.top);

	for (int i = 0; i < size; i++)
	{
		printf("%d : %d \n", i, C.top.number[i]);
	}

	printf("\nCpu finished...\n");
}

__global__ void test()
{
	constexpr int size = 50;

	myNumLib::precisionNumber A = myNumLib::precisionNumber::devicePrecisionNumberConstructor(size);
	myNumLib::precisionNumber B = myNumLib::precisionNumber::devicePrecisionNumberConstructor(size);
	myNumLib::precisionNumber C = myNumLib::precisionNumber::devicePrecisionNumberConstructor(size);

	for (size_t i = 0; i < size; i++)
	{
		A.top.number[i] = 1;
		B.top.number[i] = 2;
	}

	C.top = myNumLib::bigInt::deviceAdd(A.top, B.top);

	for (int i = 0; i < size; i++)
	{
		printf("%d : %d \n", i, C.top.number[i]);
	}

	printf("\nKernel finished...\n");
}

int main()
{
	globals Global;
	Global.clean();
	
	printf("Do you want to accelerate computing with Cuda compatible Gpu?\n[0] - No\n[1] - Yes\n");
	while (!(sf::Keyboard::isKeyPressed(sf::Keyboard::Num0) || sf::Keyboard::isKeyPressed(sf::Keyboard::Num1)))
	{
		if (sf::Keyboard::isKeyPressed(sf::Keyboard::Num1))
		{
			Global.GpuAcceleration = true;
		}
	}
	printf("Gpu acceleration: %d \n", Global.GpuAcceleration);


	//vector<int> te = { 1,2,3,4 };
	cpTest();
	//test<< <1, 1 >> > ();

	//return 0;
	cout << "Resolution: " << Global.HEIGHT << "X" << Global.WIDTH << endl;

	sf::RenderWindow window(sf::VideoMode(Global.WIDTH, Global.HEIGHT), "Mandelbrot Set", sf::Style::Titlebar | sf::Style::Close);

	sf::Texture texture;
	texture.create(Global.WIDTH, Global.HEIGHT);

	sf::Sprite sprite(texture);

	variables vars;

	positions cords(Global.WIDTH, Global.HEIGHT);

	threadsHandling Threads(&cords, &Global);

	thread Sc_update(updatePixels_forThread, &Global);

	screenText screentext;


	while (window.isOpen())
	{
		sf::Event event;
		while (window.pollEvent(event))
		{
			if (event.type == sf::Event::Closed)
			{
				window.close();
			}
		}

		vars.variables_update(window);
		if (vars.MouseVars.left_button_down && !vars.ScreenVars.after_grab && vars.ScreenVars.has_focus)
		{
			//paint(Global.screen, vars, Global.WIDTH, Global.HEIGHT);

			vars.ScreenVars.grab_point.set_point(vars.MouseVars.mousePosition.x, vars.MouseVars.mousePosition.y);
			vars.ScreenVars.after_grab = true;

			vars.ScreenVars.lastposition.set(vars.MouseVars.mousePosition.x, vars.MouseVars.mousePosition.y);

		}
		if (vars.MouseVars.left_button_down && vars.ScreenVars.after_grab && vars.ScreenVars.has_focus)
		{


			if (pointsDistance(vars.ScreenVars.lastposition, intPoint(vars.MouseVars.mousePosition.x, vars.MouseVars.mousePosition.y), 30))
			{
				vars.ScreenVars.lastposition.set(vars.MouseVars.mousePosition.x, vars.MouseVars.mousePosition.y);

				Threads.killAll(&Global);

				sprite.setPosition(0, 0);

				int dx = (int)(vars.MouseVars.mousePosition.x - vars.ScreenVars.grab_point.x);
				int dy = (int)(vars.MouseVars.mousePosition.y - vars.ScreenVars.grab_point.y);

				Global.Pend = true;
				Sc_update.join();
				Global.Pend = false;

				moveScreen(dx, dy, &Global);
				cords.recalculate(dx, dy);

				Sc_update = thread(updatePixels_forThread, &Global);

				Threads.start(&cords, &Global);

				vars.ScreenVars.grab_point.set_point(vars.MouseVars.mousePosition.x, vars.MouseVars.mousePosition.y);
			}
			else
			{
				sprite.setPosition((int)(vars.MouseVars.mousePosition.x - vars.ScreenVars.grab_point.x), (int)(vars.MouseVars.mousePosition.y - vars.ScreenVars.grab_point.y));
			}


		}

		if (!vars.MouseVars.left_button_down && vars.ScreenVars.after_grab && vars.ScreenVars.has_focus)
		{
			Threads.killAll(&Global);

			vars.ScreenVars.after_grab = false;

			sprite.setPosition(0, 0);

			int dx = (int)(vars.MouseVars.mousePosition.x - vars.ScreenVars.grab_point.x);
			int dy = (int)(vars.MouseVars.mousePosition.y - vars.ScreenVars.grab_point.y);

			Global.Pend = true;
			Sc_update.join();
			Global.Pend = false;

			moveScreen(dx, dy, &Global);
			cords.recalculate(dx, dy);

			Sc_update = thread(updatePixels_forThread, &Global);

			Threads.start(&cords, &Global);
		}


		if ((int)(event.mouseWheelScroll.delta) > 0 && vars.ScreenVars.has_focus && (vars.MouseVars.mousePosition.x >= 0 && vars.MouseVars.mousePosition.x < Global.WIDTH && vars.MouseVars.mousePosition.y >= 0 && vars.MouseVars.mousePosition.y < Global.HEIGHT))
		{
			Threads.killAll(&Global);

			Global.Pend = true;
			Sc_update.join();
			Global.Pend = false;

			//cleanScreen(&Global);
			zoom_in(&Global, &vars);

			cords.zoom_in(vars.MouseVars.mousePosition.x, vars.MouseVars.mousePosition.y);

			cout << "Zin: " << cords.top_left.x << " : " << cords.top_left.y << endl;

			Threads.start(&cords, &Global);

			event.mouseWheelScroll.delta = 0;

			Sc_update = thread(updatePixels_forThread, &Global);
		}
		else if ((int)(event.mouseWheelScroll.delta) < 0 && vars.ScreenVars.has_focus && (vars.MouseVars.mousePosition.x >= 0 && vars.MouseVars.mousePosition.x < Global.WIDTH && vars.MouseVars.mousePosition.y >= 0 && vars.MouseVars.mousePosition.y < Global.HEIGHT))
		{
			Threads.killAll(&Global);

			Global.Pend = true;
			Sc_update.join();
			Global.Pend = false;

			//cleanScreen(&Global);
			zoom_out(&Global, &vars);

			cords.zoom_out(vars.MouseVars.mousePosition.x, vars.MouseVars.mousePosition.y);

			cout << "Zout: " << cords.top_left.x << " : " << cords.top_left.y << endl;

			Threads.start(&cords, &Global);

			event.mouseWheelScroll.delta = 0;

			Sc_update = thread(updatePixels_forThread, &Global);
		}

		if (true)
		{
			//updatePixels(Global.screen, Global.pixels, Global.WIDTH, Global.HEIGHT, Global.max_iterations);

			Global.pixelMutex.lock();
			texture.update(Global.pixels);
			Global.pixelMutex.unlock();

			window.clear();

			window.draw(sprite);
			screentext.refresh(vars.MouseVars.mousePosition.x, vars.MouseVars.mousePosition.y, cords, Global, window);
		}

		window.display();

		if (vars.MouseVars.right_button_down)
		{

			//cout << Global.screen[vars.MouseVars.mousePosition.x + Global.WIDTH * vars.MouseVars.mousePosition.y] << endl;


		}

	}
	end(Global, 0, Threads, &Sc_update);
}

